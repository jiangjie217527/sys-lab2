#include<stdio.h>
#include<hip/hip_runtime.h>
#include <time.h>

#define M 1024
#define N 1024
#define K 1024
#define THREAD_PRE_BLOCK 32 


__global__ void gemm(int*a, int *b, int *c){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    c[i*M+j]=0;
    for(int k=0;k<N;k++){
        c[i*M+j]+=a[i*M+j]+b[i*M+j];
    }
}

int main(){
    int *a, *b, *c, *c_cmp;
    a = (int*) malloc(M * K * sizeof(int));
    b = (int*) malloc(K * N * sizeof(int));
    c = (int*) malloc(M * N * sizeof(int));
    c_cmp = (int*) malloc(M * N * sizeof(int));
    srand((unsigned)time(NULL)); 
    for(int i = 0; i < M; i++){
        for(int j = 0; j < K; j++){
            a[i * K + j] = rand() % 100;
        }
    }
    for(int i = 0; i < K; i++){
        for(int j = 0; j < N; j++){
            b[i * N + j] = rand() % 100;
        }
        
    }
    int *a_d, *b_d, *c_d;
    hipMalloc(&a_d, M * K * sizeof(int));
    hipMalloc(&b_d, K * N * sizeof(int));
    hipMalloc(&c_d, M * N * sizeof(int));
    hipMemcpy(a_d, a, M * K * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, K * N * sizeof(int), hipMemcpyHostToDevice);

    dim3 block(THREAD_PRE_BLOCK, THREAD_PRE_BLOCK);    // block及thread的分配方式可以自己修改
    dim3 grid(M/THREAD_PRE_BLOCK, N/THREAD_PRE_BLOCK);
    gemm<<<grid, block>>>(a_d, b_d, c_d);

    hipMemcpy(c, c_d, M * N * sizeof(int), hipMemcpyDeviceToHost);
    for(int i = 0; i < M; i++){
        for(int j = 0; j < N; j++){
            for(int k = 0; k < K; k++){
                c_cmp[i * N + j] += a[i * K + k] * b[k * N + j]; 
            }
        }
    }
    bool flag = 1;
    for(int i = 0; i < M; i++){
        for(int j = 0; j < N; j++){
            if(c_cmp[i * N + j] != c[i * N + j]){
                flag = 0;
                break;
            }
        }
        if(flag==0)break;
    }
    if(flag){
        printf("result correct\n");
    }
    else{
        printf("result wrong\n");
    }
    free(a);
    free(b);
    free(c);
    free(c_cmp);
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
}
