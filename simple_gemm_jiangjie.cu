#include<stdio.h>
#include<hip/hip_runtime.h>
#include <time.h>
#define SIZE__ 4096
#define M SIZE__
#define N SIZE__
#define K SIZE__
#define THREAD_PRE_BLOCK 8 


__global__ void gemm(int*a, int *b, int *c){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(i<M&&j<N){//防止越界
	/*
    	int res=0,index_a=i*M,index_b=j;
    	for(int k=0;k<K;++k){
	    res+=a[index_a]*b[index_b];
	    ++index_a;
	    index_b+=M;
    	}
	这样计算下标不会变快
	*/ 
    	int res=0;
    	for(int k=0;k<K;++k){
	    res+=a[i*M+k]*b[k*K+j];
    	}
	c[i*M+j]=res;
    }
}

int main(){
//  freopen("log.out","w",stdout);
    int *a, *b, *c, *c_cmp;
    a = (int*) malloc(M * K * sizeof(int));
    b = (int*) malloc(K * N * sizeof(int));
    c = (int*) malloc(M * N * sizeof(int));
    c_cmp = (int*) malloc(M * N * sizeof(int));
    srand((unsigned)time(NULL)); 
    for(int i = 0; i < M; i++){
        for(int j = 0; j < K; j++){
            a[i * K + j] = rand() % 1024;
        }
    }
    for(int i = 0; i < K; i++){
        for(int j = 0; j < N; j++){
            b[i * N + j] = rand() % 1024;
        }
        
    }
    int *a_d, *b_d, *c_d;
    hipMalloc(&a_d, M * K * sizeof(int));
    hipMalloc(&b_d, K * N * sizeof(int));
    hipMalloc(&c_d, M * N * sizeof(int));
    hipMemcpy(a_d, a, M * K * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, K * N * sizeof(int), hipMemcpyHostToDevice);

    dim3 block(THREAD_PRE_BLOCK, THREAD_PRE_BLOCK);    // block及thread的分配方式可以自己修改
    dim3 grid(M/THREAD_PRE_BLOCK+1, N/THREAD_PRE_BLOCK+1); //防止M,N不能被整除
    gemm<<<grid, block>>>(a_d, b_d, c_d);
    if(SIZE__<=1024){
        hipMemcpy(c, c_d, M * N * sizeof(int), hipMemcpyDeviceToHost);
        for(int i = 0; i < M; i++){
            for(int j = 0; j < N; j++){
                for(int k = 0; k < K; k++){
                    c_cmp[i * N + j] += a[i * K + k] * b[k * N + j]; 
                }
            }
        }
        bool flag = 1;
        for(int i = 0; i < M; i++){
            for(int j = 0; j < N; j++){
                if(c_cmp[i * N + j] != c[i * N + j]){
                    flag = 0;
                    break;
                }
            }
            if(flag==0)break;
        }
        if(flag){
            printf("result correct\n");
        }
        else{
            printf("result wrong\n");
        }
    }
    else {
    	printf("no verify\n");
    }
    free(a);
    free(b);
    free(c);
    free(c_cmp);
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
}
